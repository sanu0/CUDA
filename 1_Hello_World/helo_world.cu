
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU() {
    printf("Hello from GPU!\\n");
}

int main() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}